#include "hip/hip_runtime.h"
#include "alloc.h"
#include "net_ib.h"
#include <mpi.h>
#include <unistd.h>
#define bytes 1024
#define TAG 7

// this PORT is set in nccl/standalone/misc/socket.cc ncclSocketListen line 380.
// In previous NCCL code, this port is 0 (any port). I guess NCCL exchanged the
// port and IP address in some ways, otherwise their is no way to know the port
// and address of the other side.
#define PORT 40000
// this ADDR is the IP address of mlx5_ib0 on my machine, set it to your own.
// This IP address is used as OOB(Out of Band) address, which is used to
// establish the connection and exchange the needed information like the cq and
// qp of the IB connection.
#define ADDR "172.16.1.138"

// all of the functions of ncclNetIb is non-blocking, so we need to run them in
// a loop
int ib_send()
{
    ncclDebugLogger_t logger;
    NCCLCHECK(ncclIbInit(logger));
    char *sendbuff = NULL;
    NCCLCHECK(ncclIbMalloc((void **)&sendbuff, bytes));
    for (int i = 0; i < bytes; i++) {
        sendbuff[i] = i % 47;
    }
    ncclIbHandle handle;
    handle.connectAddr.sin.sin_family = AF_INET;
    handle.connectAddr.sin.sin_port = htons(PORT);
    inet_aton(ADDR, &handle.connectAddr.sin.sin_addr);
    // this magic is used to identify if the connection is established by NCCL
    handle.magic = NCCL_SOCKET_MAGIC;
    // the sender uses ib1
    ncclIbSendComm *sendComm = NULL;
    while (sendComm == NULL) {
        NCCLCHECK(ncclIbConnect(1, &handle, (void **)&sendComm));
    }
    ibv_mr *mhandle;
    NCCLCHECK(ncclIbRegMr(sendComm, sendbuff, bytes, NCCL_PTR_HOST,
                          (void **)&mhandle));
    struct ncclIbRequest *requset = NULL;

    while (requset == NULL) {
        // the ncclIbIsend is non-blocking, so we need to run it in a loop
        NCCLCHECK(ncclIbIsend(sendComm, sendbuff, bytes, TAG, mhandle,
                              (void **)&requset));
    }
    int done = 0;
    int finished_size = 0;
    while (done == 0) {
        NCCLCHECK(ncclIbTest(requset, &done, &finished_size));
    }
    if (finished_size != bytes) {
        printf("Error: finished_size=%d\n", finished_size);
    }
}

int ib_recv()
{
    ncclDebugLogger_t logger;

    NCCLCHECK(ncclIbInit(logger));
    char *recvbuff = NULL;
    NCCLCHECK(ncclIbMalloc((void **)&recvbuff, bytes));
    ncclIbHandle handle;
    ncclIbListenComm *listenComm;
    NCCLCHECK(ncclIbListen(2, &handle, (void **)&listenComm));

    ncclIbRecvComm *recvComm = NULL;
    while (recvComm == NULL) {
        NCCLCHECK(ncclIbAccept(listenComm, (void **)&recvComm));
    }

    ibv_mr *mhandle;
    NCCLCHECK(ncclIbRegMr(recvComm, recvbuff, bytes, NCCL_PTR_HOST,
                          (void **)&mhandle));
    int size = bytes;
    int tag = TAG;
    struct ncclIbRequest *requset = NULL;
    NCCLCHECK(ncclIbIrecv(recvComm, 1, (void **)&recvbuff, &size, &tag,
                          (void **)&mhandle, (void **)&requset));
    int done = 0;
    int finished_size = 0;
    while (done == 0) {
        NCCLCHECK(ncclIbTest(requset, &done, &finished_size));
    }
    // check the recvbuff
    for (int i = 0; i < bytes; i++) {
        if (recvbuff[i] != i % 47) {
            printf("Error: recvbuff[%d]=%d\n", i, recvbuff[i]);
            return -1;
        }
    }
    printf("Success\n");
}

int ib_sendrecv_test(int world_rank)
{
    if (world_rank == 0) {
        ib_send();
    } else if (world_rank == 1) {
        ib_recv();
    }
}

int main()
{
    setbuf(stdout, NULL);

    MPI_Init(NULL, NULL);
    // Get the number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Get the rank of the process
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    setbuf(stdout, NULL);
    ib_sendrecv_test(world_rank);
    MPI_Finalize();
}